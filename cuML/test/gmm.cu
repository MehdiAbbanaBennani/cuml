#include "hmm/gmm.h"

template <typename T>
void run(magma_int_t nCl, magma_int_t nDim, magma_int_t nObs, int n_iter)
{
// declaration:
        T *dX=NULL;
        T **dX_array=NULL;

        magma_int_t lddx = magma_roundup(nDim, RUP_SIZE);

        // allocation:
        allocate(dX, lddx * nObs);
        allocate(dX_array, nObs);

// filling:
        fill_matrix_gpu(nDim, nObs, dX, lddx);

        print_matrix_device(nDim, nObs, dX, lddx, "dX");

// Batching
        split_to_batches(nObs, dX_array, dX, lddx);

// // computation:
        GMM<T> gmm = GMM<T>(nCl, nDim, nObs);
        gmm.initialize();
        gmm.fit(dX, 6);

// cleanup:
        CUDA_CHECK(hipFree(dX));
        CUDA_CHECK(hipFree(dX_array));
}


int main( int argc, char** argv )
{
        magma_init();


        magma_int_t nCl = 2;
        magma_int_t nDim = 3;
        magma_int_t nObs = 5;
        int n_iter = 10;

        run<double>(nCl, nDim, nObs, n_iter);

        magma_finalize();
        return 0;
}

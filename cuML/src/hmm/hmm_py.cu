#include "hmm/hmm.h"
#include "hmm/hmm_py.h"

void init_f32(){
        init_hmm();
}

void setup_f32(HMM<float> &hmm) {
        setup_hmm(hmm);
}

void forward_f32(HMM<float>& hmm,
                 float* dX,
                 int* len_array,
                 int nObs){

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);
        workspaceCreate(hmm);

        forward(dX, len_array, hmm, nObs, cublasHandle, queue);

        workspaceFree(hmm);

}

void backward_f32(HMM<float>& hmm,
                  float* dX,
                  int* len_array,
                  int nObs){

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);
        workspaceCreate(hmm);

        backward(dX, len_array, hmm, nObs, cublasHandle, queue);

        workspaceFree(hmm);
}

void viterbi_f32(HMM<float>& hmm,
                 float* dX,
                 int* len_array,
                 int nObs){

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);
        workspaceCreate(hmm);

        viterbi(dX, len_array, hmm, nObs, cublasHandle, queue);

        workspaceFree(hmm);
}

void em_f32(HMM<float>& hmm,
            float* dX,
            int* len_array,
            int nObs){

        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);
        workspaceCreate(hmm);

        em(dX, len_array, hmm, nObs, cublasHandle, queue);

        workspaceFree(hmm);
}

void free_f32(HMM<float> &hmm) {
        free_hmm(hmm);
}

#include "hmm/gmm.h"
#include "hmm/gmm_py.h"

namespace gmm {

void init_f32(GMM<float> &gmm,
              float *dmu, float *dsigma, float *dPis, float *dPis_inv, float *dLlhd,
              int lddx, int lddmu, int lddsigma, int lddsigma_full, int lddPis, int lddLlhd,
              float *cur_llhd, float reg_covar,
              int nCl, int nDim, int nObs){
        init(gmm,
             dmu, dsigma, dPis, dPis_inv, dLlhd,
             lddx, lddmu, lddsigma, lddsigma_full, lddPis, lddLlhd,
             cur_llhd, reg_covar,
             nCl, nDim, nObs);
}


void update_llhd_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        update_llhd(dX, gmm, cublasHandle);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_rhos_f32(GMM<float>& gmm, float* dX){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_rhos(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_mus_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_mus(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_sigmas_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_sigmas(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_pis_f32(GMM<float>& gmm){
        update_pis(gmm);
}

void setup_f32(GMM<float> &gmm) {
        setup(gmm);
}

void free_f32(GMM<float> &gmm) {
        free(gmm);
}

}

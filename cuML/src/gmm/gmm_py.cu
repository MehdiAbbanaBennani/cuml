/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gmm/gmm.h"
#include "gmm/gmm_py.h"

namespace gmm {

void init_f32(GMM<float> &gmm,
              float *dmu, float *dsigma, float *dPis, float *dPis_inv, float *dB,
              int lddx, int lddmu, int lddsigma, int lddsigma_full, int lddPis, int lddLlhd,
              float *cur_llhd, float reg_covar,
              int nCl, int nDim, int nObs){
        init(gmm,
             dmu, dsigma, dPis, dPis_inv, dB,
             lddx, lddmu, lddsigma, lddsigma_full, lddPis, lddLlhd,
             cur_llhd, reg_covar,
             nCl, nDim, nObs);
}

void compute_lbow_f32(GMM<float>& gmm){
        compute_lbow(gmm);
}

void update_llhd_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        update_llhd(dX, gmm, cublasHandle);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_rhos_f32(GMM<float>& gmm, float* dX){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_rhos(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_mus_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_mus(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_sigmas_f32(float* dX, GMM<float>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_sigmas(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_pis_f32(GMM<float>& gmm){
        update_pis(gmm);
}

void setup_f32(GMM<float> &gmm) {
        setup(gmm);
}

void free_f32(GMM<float> &gmm) {
        free(gmm);
}


void init_f64(GMM<double> &gmm,
              double *dmu, double *dsigma, double *dPis, double *dPis_inv, double *dLlhd,
              int lddx, int lddmu, int lddsigma, int lddsigma_full, int lddPis, int lddLlhd,
              double *cur_llhd, double reg_covar,
              int nCl, int nDim, int nObs){
        init(gmm,
             dmu, dsigma, dPis, dPis_inv, dLlhd,
             lddx, lddmu, lddsigma, lddsigma_full, lddPis, lddLlhd,
             cur_llhd, reg_covar,
             nCl, nDim, nObs);
}

void compute_lbow_f64(GMM<double>& gmm){
        compute_lbow(gmm);
}

void update_llhd_f64(double* dX, GMM<double>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        update_llhd(dX, gmm, cublasHandle);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_rhos_f64(GMM<double>& gmm, double* dX){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_rhos(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_mus_f64(double* dX, GMM<double>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_mus(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_sigmas_f64(double* dX, GMM<double>& gmm){
        hipblasHandle_t cublasHandle;
        CUBLAS_CHECK(hipblasCreate(&cublasHandle));

        int device = 0;
        magma_queue_t queue;
        magma_queue_create(device, &queue);

        update_sigmas(dX, gmm, cublasHandle, queue);

        CUBLAS_CHECK(hipblasDestroy(cublasHandle));
}

void update_pis_f64(GMM<double>& gmm){
        update_pis(gmm);
}

void setup_f64(GMM<double> &gmm) {
        setup(gmm);
}

void free_f64(GMM<double> &gmm) {
        free(gmm);
}


}

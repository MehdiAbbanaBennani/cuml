#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include "hmm/determinant.h"
#include "hmm/magma/determinant.h"


template <typename T>
void run_cuda_det(int batchCount, T** dA_array, T* dDet_cusolver, bool is_hermitian){
        T **A_array, *Det_cusolver;
        A_array = (T **)malloc(sizeof(T*) * batchCount);
        Det_cusolver = (T *)malloc(sizeof(T) * batchCount);

        CUSOLVER_CHECK(hipsolverDnCreate(&cusolverHandle));
        Det = Determinant<T>(nDim, &cusolverHandle);

        updateHost(A_array, dA_array, batchCount);

        for(int bId = 0; bId < batchCount; bId++) {
                Det_cusolver[bId] = Det.compute(A_array[bId], is_hermitian);
        }

        updateDevice(dDet_cusolver, Det_cusolver, batchCount)

        CUDA_CHECK(hipFree(A_array));
        CUDA_CHECK(hipFree(Det_cusolver));
        CUSOLVER_CHECK(hipsolverDnDestroy(cusolverHandle));
        Det->TearDown();
}

template <typename T>
T run(magma_int_t n, magma_int_t batchCount)
{
// declaration:
        T **dA_array=NULL, *dDet_cusolver=NULL, *dDet_magma=NULL;
        magma_int_t ldda = magma_roundup(n, RUP_SIZE); // round up to multiple of 32 for best GPU performance
        T *error_d, error = 0;

// allocation:
        allocate_pointer_array(dA_array, ldda * n, batchCount);
        allocate(dDet_magma, batchCount);
        allocate(error, 1);

        int device = 0;  // CUDA device ID
        magma_queue_t queue;
        magma_queue_create(device, &queue);

// filling:
        fill_matrix_gpu_batched(n, n, batchCount, dA_array, ldda);

// computation magma :
        print_matrix_batched(n, n, batchCount, dA_array, ldda, "A array");

        det_batched(n, dA_array, ldda, dDet_magma, batchCount, queue);

        print_matrix_device(n, 1, dDet_magma, n, "det array");

// computation cusolver :
        run_cuda_det(batchCount, dA_array, dDet_magma, is_hermitian);

// Error
        meanSquaredError(error_d, dDet_cusolver, dDet_magma, batchCount);
        updateHost(&error, error_d, 1);

// cleanup:
        free_pointer_array(dA_array, batchCount);
        CUDA_CHECK(hipFree(dDet_magma));
        CUDA_CHECK(hipFree(dDet_cusolver));
        CUDA_CHECK(hipFree(error_d));

        return error;
}


template <typename T>
struct DeterminantInputs {
        T tolerance;
        bool is_hermitian;
        magma_int_t n, batchCount;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const DeterminantInputs<T>& dims) {
        return os;
}

template <typename T>
class DeterminantTest : public ::testing::TestWithParam<DeterminantInputs<T> > {
protected:
void SetUp() override {
        params = ::testing::TestWithParam<DeterminantInputs<T> >::GetParam();
        tolerance = params.tolerance;

        magma_init();
        error = run<T>(params.n, params.batchCount);
        magma_finalize();
}

protected:
DeterminantInputs<T> params;
T error, tolerance;
};

const std::vector<DeterminantInputs<float> > DeterminantInputsf2 = {
        {0.000001f, true, 2, 4}
};

const std::vector<DeterminantInputs<double> > DeterminantInputsd2 = {
        {0.000001, true, 2, 4}
};


typedef DeterminantTest<float> DeterminantTestF;
TEST_P(DeterminantTestF, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

typedef DeterminantTest<double> DeterminantTestD;
TEST_P(DeterminantTestD, Result){
        EXPECT_LT(error, tolerance) << " error out of tol.";
}

INSTANTIATE_TEST_CASE_P(DeterminantTests, DeterminantTestF,
                        ::testing::ValuesIn(DeterminantInputsf2));

INSTANTIATE_TEST_CASE_P(DeterminantTests, DeterminantTestD,
                        ::testing::ValuesIn(DeterminantInputsd2));
